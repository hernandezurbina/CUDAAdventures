#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cstring>


#include "common.h"


__global__ void sum_array_gpu(int *a, int *b, int *c, int size){
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) {
    c[gid] = a[gid] + b[gid];
  }
}

void sum_array_cpu(int *a, int *b, int *c, int size){
  for(int i =0; i < size; i++){
    c[i] = a[i] + b[i];
  }
}

int main(){

  int size = 10000;
  int block_size = 128;
  int NO_BYTES = sizeof(int) * size;
  hipError_t error;

  // host pointers
  int *h_a, *h_b, *h_c, *gpu_results;

  h_a = (int *) malloc(NO_BYTES);
  h_b = (int *) malloc(NO_BYTES);
  h_c = (int *) malloc(NO_BYTES);
  gpu_results = (int *) malloc(NO_BYTES);

  time_t t;
  srand((unsigned) time(&t));
  for(int i = 1; i < size; i++){
    h_a[i] = (int) (rand() & 0xFF);
    h_b[i] = (int) (rand() & 0xFF);
  }

  clock_t cpu_start, cpu_end;
  cpu_start = clock();
  sum_array_cpu(h_a, h_b, h_c, size);
  cpu_end = clock();

  // device pointers
  int *d_a, *d_b, *d_c;

  error = hipMalloc((void **)&d_a, NO_BYTES);
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s\n", hipGetErrorString(error));
  }

  error = hipMalloc((void **)&d_b, NO_BYTES);
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s\n", hipGetErrorString(error));
  }

  error = hipMalloc((void **)&d_c, NO_BYTES);
  if (error != hipSuccess) {
    fprintf(stderr, "Error: %s\n", hipGetErrorString(error));
  }

  clock_t htod_start, htod_end;
  htod_start = clock();
  hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);
  htod_end = clock();

  dim3 block(block_size);
  dim3 grid((size/block.x) + 1);

  clock_t gpu_start, gpu_end;
  gpu_start = clock();
  sum_array_gpu <<<block, grid>>>(d_a, d_b, d_c, size);
  hipDeviceSynchronize();
  gpu_end = clock();

  clock_t dtoh_start, dtoh_end;
  dtoh_start = clock();
  hipMemcpy(gpu_results, d_c, NO_BYTES, hipMemcpyDeviceToHost);
  dtoh_end = clock();

  compare_arrays(gpu_results, h_c, size);


  printf("Sum array CPU exec time: %4.6f\n",(double)((double)(cpu_end - cpu_start)/CLOCKS_PER_SEC));
  printf("Sum array GPU exec time: %4.6f\n",(double)((double)(gpu_end - gpu_start)/CLOCKS_PER_SEC));
  printf("H2D transfer time: %4.6f\n",(double)((double)(htod_end - htod_start)/CLOCKS_PER_SEC));
  printf("D2H transfer time: %4.6f\n",(double)((double)(dtoh_end - dtoh_start)/CLOCKS_PER_SEC));

  printf("Total GPU exec time: %4.6f\n", (double)((double)(dtoh_end - htod_start)/CLOCKS_PER_SEC));

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(gpu_results);

  return 0;
}
