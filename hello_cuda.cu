#include "hip/hip_runtime.h"


#include <stdio.h>


// DEVICE CODE:
// Kernel:
__global__ void hello_cuda(){
	printf("Hello CUDA world!\n");
}

// HOST CODE
int main(){

	// launching kernel:
	hello_cuda<<<1,20>>>();
	hipDeviceSynchronize();
	hipDeviceReset();

	return 0;
}