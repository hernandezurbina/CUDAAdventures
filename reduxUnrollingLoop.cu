#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void reduxUnrollingBlocks2(int *input, int *temp, int size){
  int tid = threadIdx.x;
  int BLOCK_OFFSET = blockIdx.x * blockDim.x * 2;
  int index = BLOCK_OFFSET + tid;
  int *i_data = input + BLOCK_OFFSET;

  if((index + blockDim.x) < size) {
    input[index] += input[index + blockDim.x];
  }
  __syncthreads();

  for(int offset = blockDim.x/2; offset > 0; offset = offset /2){
    if(tid < offset) {
      temp[blockIdx.x] = i_data[0];
    }
    __syncthreads();
  }
  if(tid == 0) {
    temp[blockIdx.x] = i_data[0];
  }

}

__global__ void reduxUnrollingBlocks4(int *input, int *temp, int size){
  int tid = threadIdx.x;
  int BLOCK_OFFSET = blockIdx.x * blockDim.x * 4;
  int index = BLOCK_OFFSET + tid;
  int *i_data = input + BLOCK_OFFSET;

  if((index + 3 * blockDim.x) < size) {
    int a1 = input[index];
    int a2 = input[index + blockDim.x];
    int a3 = input[index + 2 * blockDim.x];
    int a4 = input[index + 3 * blockDim.x];
    input[index] = a1 + a2 + a3 + a4;
  }
  __syncthreads();

  for(int offset = blockDim.x/2; offset > 0; offset = offset /2){
    if(tid < offset) {
      temp[blockIdx.x] = i_data[0];
    }
    __syncthreads();
  }
  if(tid == 0) {
    temp[blockIdx.x] = i_data[0];
  }

}

int main(int argc, char** argv){
  int size = 1 << 27;
  int byte_size = size * sizeof(int);

  int block_size = 128;

  int *h_input, *h_ref;
  h_input = (int *) malloc(byte_size);

  initialize(h_input, size, INIT_RANDOM);

  // get reduction result from CPU
  int cpu_result = reduction_cpu(h_input, size);

  dim3 block(block_size);
  dim3 grid((size / block_size)/2);

  printf("Kernel launch params:\n grid.x: %d, block.x: %d\n", grid.x, block.x);

  int temp_array_byte_size = sizeof(int) * grid.x;
  h_ref = (int *) malloc(temp_array_byte_size);

  int *d_input, *d_temp;

  hipMalloc((void **)&d_input, byte_size);
  hipMalloc((void **)&d_temp, temp_array_byte_size);

  hipMemset(d_temp, 0, temp_array_byte_size);
  hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

  reduxUnrollingBlocks2 <<<grid, block>>>(d_input, d_temp, size);

  hipDeviceSynchronize();
  hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost);

  int gpu_result = 0;
  for(int i = 0; i < grid.x; i++) {
    gpu_result += h_ref[i];
  }

  compare_results(gpu_result, cpu_result);

  free(h_ref);
  free(h_input);

  hipFree(d_temp);
  hipFree(d_input);

  hipDeviceReset();
  return 0;

}
