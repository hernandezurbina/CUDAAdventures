#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void reduction_neighbor_pair(int *input, int *temp, int size){

}

int main(int argc, char **argv){

  printf("Running neighboring reduction pair kernel\n");

  int size = 1 << 27;
  int byte_size = size * sizeof(int);

  int block_size = 128;

  int *h_input, *h_ref;
  h_input = (int *) malloc(byte_size);

  initialize(h_input, size, INIT_RANDOM);

  // get reduction result from CPU
  int cpu_result = reduction_cpu(h_input, size);

  dim3 block(block_size);
  dim3 grid(size/block.x);

  printf("Kernel launch params:\n grid.x: %d, block.x: %d\n", grid.x, block.x);

  int temp_array_byte_size = sizeof(int) * grid.x;
  h_ref = (int *) malloc(temp_array_byte_size);

  int *d_input, *d_temp;

  hipMalloc((void **)&d_input, byte_size);
  hipMalloc((void **)&d_temp, temp_array_byte_size);

  hipMemset(d_temp, 0, temp_array_byte_size);

  hipDeviceReset();
  return 0;
}
