#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void unique_index_calc_threadIdx(int *input){
	// receives a pointer to an int
	int tid = threadIdx.x;
	printf("threadIdx: %d, value: %d\n", tid, input[tid]);
}

int main(){

	int array_size = 8;
	int array_bit_size = sizeof(int) * array_size;
	int h_data[] = {23, 9, 4, 53, 65, 12, 1, 33};

	for(int i = 0; i < array_size; i++){
		printf("%d ", h_data[i]);
	}
	printf("\n\n");

	int *d_data;

	hipMalloc((void **)&d_data, array_bit_size);
	hipMemcpy(d_data, h_data, array_bit_size, hipMemcpyHostToDevice);

	dim3 block(8);
	dim3 grid(1);

	unique_index_calc_threadIdx <<<grid, block>>>(d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}