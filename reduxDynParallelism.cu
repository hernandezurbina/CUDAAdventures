#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "common.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void gpuRecursiveRedux(int *g_idata, int *g_odata, unsigned int isize){
  int tid = threadIdx.x;
  int *idata = g_idata + blockIdx.x * blockDim.x;
  int *odata = &g_odata[blockIdx.x];

  if(isize == 2 && tid == 0){
    g_odata[blockIdx.x] = idata[0] + idata[1];
    return;
  }

  int istride = isize >> 1;
  if(istride > 1 && tid < istride) {
    idata[tid] += idata[tid + istride];
  }

  __syncthreads();

  if(tid == 0){
    gpuRecursiveRedux<<<1, istride>>>(idata, odata, istride);
    hipDeviceSynchronize();
  }
  __syncthreads();
}

int main(){
  int size = 1 << 27;
  int byte_size = size * sizeof(int);

  int block_size = 128;

  int *h_input, *h_ref;
  h_input = (int *) malloc(byte_size);

  initialize(h_input, size, INIT_RANDOM);

  // get reduction result from CPU
  int cpu_result = reduction_cpu(h_input, size);

  dim3 block(block_size);
  dim3 grid(size/block.x);

  printf("Kernel launch params:\n grid.x: %d, block.x: %d\n", grid.x, block.x);

  int temp_array_byte_size = sizeof(int) * grid.x;
  h_ref = (int *) malloc(temp_array_byte_size);

  int *d_input, *d_temp;

  hipMalloc((void **)&d_input, byte_size);
  hipMalloc((void **)&d_temp, temp_array_byte_size);

  hipMemset(d_temp, 0, temp_array_byte_size);
  hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

  gpuRecursiveRedux<<<grid, block>>>(d_input, d_temp, size);

  hipDeviceSynchronize();
  hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost);

  int gpu_result = 0;
  for(int i = 0; i < grid.x; i++) {
    gpu_result += h_ref[i];
  }

  compare_results(gpu_result, cpu_result);

  free(h_ref);
  free(h_input);

  hipFree(d_temp);
  hipFree(d_input);

  hipDeviceReset();
  return 0;

}
