#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "common.h"

__global__ void simple_kernel(){
  printf("Hello from kernel!\n");
}

int main(){
  simple_kernel <<<1, 1>>>();
  simple_kernel <<<1, 1>>>();
  simple_kernel <<<1, 1>>>();

  hipDeviceSynchronize();
  hipDeviceReset();
  
  return 0;
}
