#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "common.h"

__global__ void simple_kernel(){
  printf("Hello from kernel!\n");
}

int main(){

  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);

  if(deviceProp.concurrentKernels == 0){
    printf("GPU doesn't support concurrent execution\n");
    printf("Kernel exec will be serialized\n");
  }

  hipStream_t str1, str2, str3;
  hipStreamCreate(&str1);
  hipStreamCreate(&str2);
  hipStreamCreate(&str3);

  simple_kernel <<<1, 1, 0, str1>>>();
  simple_kernel <<<1, 1, 0, str2>>>();
  simple_kernel <<<1, 1, 0, str3>>>();

  hipStreamDestroy(str1);
  hipStreamDestroy(str2);
  hipStreamDestroy(str3);

  hipDeviceSynchronize();
  hipDeviceReset();

  return 0;
}
