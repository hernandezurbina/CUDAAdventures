#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
// #include <time.h>
// #include <cstring>


#include "common.h"


__global__ void stream_test_modified(int *in, int *out, int size){
  int gid = blockIdx.x * blockDim.x + threadIdx.x;

  if (gid < size) {
    for(int i = 0; i < 25; i++){
      out[gid] = in[gid] + (in[gid] - 1) * (gid % 10);
    }
  }
}


int main(){

  int size = 1 << 18;
  int byte_size = size * sizeof(int);

  // host pointers
  int *h_in, *h_ref;

  hipHostMalloc((void **)&h_in, byte_size);
  hipHostMalloc((void **)&h_ref, byte_size);

  h_in = (int *) malloc(byte_size);
  h_ref = (int *) malloc(byte_size);
  initialize(h_in, INIT_RANDOM);

  // device pointers
  int *d_in, *d_out;

  hipMalloc((void **)&d_in, byte_size);
  hipMalloc((void **)&d_out, byte_size);

  hipStream_t str;
  hipStreamCreate(&str);

  hipMemcpyAsync(d_in, h_in, byte_size, hipMemcpyHostToDevice, str);

  dim3 block(128);
  dim3 grid(size/block.x);

  stream_test_modified <<<grid, block, 0, str>>>(d_in, d_out, size);
  hipDeviceSynchronize();

  hipMemcpyAsync(h_ref, d_out, byte_size, hipMemcpyDeviceToHost, str);
  hipStreamSynchronize(str);
  hipStreamDestroy(str);

  hipDeviceReset();
  return 0;
}
