#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void unique_index_calc_threadIdx(int *input){
	// receives a pointer to an int
	int tid = threadIdx.x;
	printf("threadIdx: %d, value: %d\n", tid, input[tid]);
}

__global__ void unique_gid_calculation(int *input){
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;

	printf("blockIdx.x: %d, threadIdx.x: %d, gid: %d, value: %d\n", blockIdx.x, tid, gid, input[gid]);
}

int main(){

	int array_size = 16;
	int array_bit_size = sizeof(int) * array_size;
	int h_data[] = {23, 9, 4, 53, 65, 12, 1, 33, 3, 92, 41, 54, 68, 11, 45, 21};

	for(int i = 0; i < array_size; i++){
		printf("%d ", h_data[i]);
	}
	printf("\n\n");

	int *d_data;

	hipMalloc((void **)&d_data, array_bit_size);
	hipMemcpy(d_data, h_data, array_bit_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(4);

	//unique_index_calc_threadIdx <<<grid, block>>>(d_data);
	unique_gid_calculation <<<grid, block>>>(d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}